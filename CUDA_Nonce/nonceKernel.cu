#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>

#include "config.h"
#include "fmt.h"

// SHA1 Implementation
struct CUDA_SHA1_CTX {
	BYTE data[64];
	WORD datalen;
	LONG bitlen;
	WORD state[5];
	WORD k[4];
};

#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

__constant__ CUDA_SHA1_CTX HASHED_INPUT;

__host__ __device__ void cuda_sha1_transform(CUDA_SHA1_CTX* ctx, const BYTE data[])
{
	WORD a, b, c, d, e, i, j, t, m[80];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + (data[j + 3]);
	for (; i < 80; ++i) {
		m[i] = (m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16]);
		m[i] = (m[i] << 1) | (m[i] >> 31);
	}

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	for (i = 0; i < 20; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + ctx->k[0] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 40; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[1] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 60; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d)) + e + ctx->k[2] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 80; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[3] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}

void sha1_init(CUDA_SHA1_CTX* ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;
	ctx->k[0] = 0x5a827999;
	ctx->k[1] = 0x6ed9eba1;
	ctx->k[2] = 0x8f1bbcdc;
	ctx->k[3] = 0xca62c1d6;
}

__host__ __device__ void cuda_sha1_update(CUDA_SHA1_CTX* ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			cuda_sha1_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void cuda_sha1_final(CUDA_SHA1_CTX* ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_sha1_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	cuda_sha1_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and MD uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

__device__ BYTE* d_strcpy(BYTE* dest, BYTE* src) {
	int i = 0;

	do {
		dest[i] = src[i];
	} while (src[i++] != 0);

	return dest;
}

__device__ void d_reverse(BYTE str[], int size)
{
	int start = 0;
	int end = size - 1;
	while (start < end) {
		char h = *(str + start), t = *(str + end);
		*(str + start) = t;
		*(str + end) = h;
		start++;
		end--;
	}
}

__device__ BYTE* d_itob(size_t num, BYTE* str, int* size)
{
	int i = 0;

	/* Handle 0 explicitely, otherwise empty string is printed for 0 */
	if (num == 0) {
		str[i++] = '0';
		return str;
	}

	// Process individual digits 
	while (num != 0) {
		int rem = num % 10;
		str[i++] = (rem > 9) ? (rem - 10) + 'a' : rem + '0';
		num = num / 10;
	}

	// Reverse the string 
	d_reverse(str, i);
	*size = i;

	return str;
}

__device__ BYTE* d_strcat(BYTE* dest, BYTE* src) {
	int i = 0;

	while (dest[i] != 0) {
		i++;
	}
	d_strcpy(dest + i, src);

	return dest;
}

__device__ void makedigits(BYTE x, BYTE(&digits)[2])
{
	BYTE d0 = x / 16;
	digits[1] = x - d0 * 16;
	BYTE d1 = d0 / 16;
	digits[0] = d0 - d1 * 16;
}

__device__ void makehex(BYTE(&digits)[2], char(&hex)[2])
{
	for (int i = 0; i < 2; ++i) {
		if (digits[i] < 10) {
			hex[i] = '0' + digits[i];
		}
		else {
			hex[i] = 'a' + (digits[i] - 10);
		}
	}
}

// The kernel
__global__ void find_nonce(size_t* result, BYTE* hash, bool* found, size_t stride) {
	// Copy the hashed input to the thread's local memory
	CUDA_SHA1_CTX thread_ctx;
	thread_ctx.bitlen = HASHED_INPUT.bitlen;
	thread_ctx.datalen = HASHED_INPUT.datalen;
	thread_ctx.state[0] = HASHED_INPUT.state[0];
	thread_ctx.state[1] = HASHED_INPUT.state[1];
	thread_ctx.state[2] = HASHED_INPUT.state[2];
	thread_ctx.state[3] = HASHED_INPUT.state[3];
	thread_ctx.state[4] = HASHED_INPUT.state[4];
	thread_ctx.k[0] = HASHED_INPUT.k[0];
	thread_ctx.k[1] = HASHED_INPUT.k[1];
	thread_ctx.k[2] = HASHED_INPUT.k[2];
	thread_ctx.k[3] = HASHED_INPUT.k[3];
	d_strcpy(thread_ctx.data, HASHED_INPUT.data);


	BYTE checksum[SHA_SIZE];
	memset(checksum, 0x0, SHA_SIZE);

	unsigned int thread = blockIdx.x * blockDim.x + threadIdx.x;
	size_t nonce_source = thread + stride;

	// Prepare the input
	int nonce_size = 0;
	BYTE nonce[SHA_SIZE];

	d_itob(nonce_source, nonce, &nonce_size);

	cuda_sha1_update(&thread_ctx, nonce, nonce_size);
	cuda_sha1_final(&thread_ctx, checksum);

	bool suffix_matches = true;
	for (int i = 0; i < ZEROS_TO_FIND; i++) {
		if (checksum[SHA_SIZE - i - 1] != 0x0) {
			suffix_matches = false;
			break;
		}
	}

	if (suffix_matches) {
		*found = true;
		*result = nonce_source;
		d_strcpy(hash, checksum);
	}
}

void get_optimal_sizes(int* grid_size, int* block_size)
{
	hipDeviceProp_t deviceProp;
	int min_grid_size;

	if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) {
		fprintf(stderr, "hipGetDeviceProperties failed!");
		return;
	}

	hipOccupancyMaxPotentialBlockSize(&min_grid_size, block_size, find_nonce, 0, 0);

	// Calculate the maximum grid size based on the number of multiprocessors
	*grid_size = (deviceProp.multiProcessorCount * min_grid_size);

	// Ensure the grid size does not exceed the maximum grid dimensions
	if (*grid_size > deviceProp.maxGridSize[0]) {
		*grid_size = deviceProp.maxGridSize[0];
	}

	// Ensure the block size does not exceed the maximum threads per block
	if (*block_size > deviceProp.maxThreadsPerBlock) {
		*block_size = deviceProp.maxThreadsPerBlock;
	}
	//*grid_size = 128;
	//*block_size = 128;
	printf("Optimal grid size: %d, block size: %d\n", *grid_size, *block_size);
}


int main(int argc, char** argv) {
	bool h_found = false;
	size_t h_nonce = 0;
	size_t nonce_size = sizeof(size_t);
	int grid_size;
	int block_size;

	size_t i = 0;
	size_t stride = 0;
	size_t th_count = 0;

	struct timeb start, end;
	double seconds = 0;

	hipError_t status = hipSuccess;

	get_optimal_sizes(&grid_size, &block_size);

	// Initialize the input data
	BYTE* h_digest = (BYTE*)malloc(SHA_SIZE);
	memset(h_digest, 0, SHA_SIZE);


	// Compute the SHA-1 of the input buffer on the host
	CUDA_SHA1_CTX h_ctx;
	sha1_init(&h_ctx);
	cuda_sha1_update(&h_ctx, (BYTE*)BUFFER, BUFFER_SIZE);
	fprintf(stdout, "size of data: %d\n", h_ctx.datalen);
	 
	// Copy the hashed input to the device's constant memory
	status = hipMemcpyToSymbol(HIP_SYMBOL(HASHED_INPUT), &h_ctx, sizeof(CUDA_SHA1_CTX));
	if (hipSuccess != status) {
		fprintf(stderr, "hipMemcpyToSymbol failed! Error: %s", hipGetErrorString(status));
		goto Error;
	}


	// Initialize the device variables
	size_t* d_nonce;
	bool* d_found;
	BYTE* d_digest;
	status = hipMalloc((void**)&d_nonce, nonce_size);
	if (hipSuccess != status) {
		fprintf(stderr, "hipMalloc failed! Error: %s", hipGetErrorString(status));
		goto Error;
	}
	status = hipMalloc((void**)&d_digest, SHA_SIZE);
	if (hipSuccess != status) {
		fprintf(stderr, "hipMalloc failed! Error: %s", hipGetErrorString(status));
		goto Error;
	}
	status = hipMalloc((void**)&d_found, sizeof(bool));
	if (hipSuccess != status) {
		fprintf(stderr, "hipMalloc failed! Error: %s", hipGetErrorString(status));
		goto Error;
	}

	status = hipMemcpy(d_found, &h_found, sizeof(bool), hipMemcpyHostToDevice);
	if (hipSuccess != status) {
		fprintf(stderr, "hipMemcpy failed! Error: %s", hipGetErrorString(status));
		goto Error;
	}


	fprintf(stdout, "Starting the search kernel with grid size %d and block size %d\n", grid_size, block_size);
	// Start the timer
	ftime(&start);

	th_count = grid_size * block_size;
	do {
		find_nonce <<<grid_size, block_size>>> (d_nonce, d_digest, d_found, stride);
		status = hipGetLastError();
		if (hipSuccess != status) {
			fprintf(stderr, "Failed to launch the kernel! Error: %s", hipGetErrorString(status));

			goto Error;
		}

		status = hipDeviceSynchronize();
		if (hipSuccess != status) {
			fprintf(stderr, "hipDeviceSynchronize failed! Error: %s", hipGetErrorString(status));
			goto Error;
		}

		status = hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
		if (hipSuccess != status) {
			fprintf(stderr, "Failed to copy the found bool back to host! Error: %s", hipGetErrorString(status));
			goto Error;
		}

		stride += th_count;
		i++;
	} while (!h_found && i <= MAX_ITERATIONS);


	// Copy the data back to the host
	status = hipMemcpy(h_digest, d_digest, SHA_SIZE, hipMemcpyDeviceToHost);
	if (hipSuccess != status) {
		fprintf(stderr, "Failed to copy the resulting hash back to host. Error: %s", hipGetErrorString(status));
		goto Error;
	}
	status = hipMemcpy(&h_nonce, d_nonce, nonce_size, hipMemcpyDeviceToHost);
	if (hipSuccess != status) {
		fprintf(stderr, "Failed to copy the found nonce back to host. Error: %s", hipGetErrorString(status));
		goto Error;
	}

	// Stop the timer
	ftime(&end);
	seconds = end.time - start.time + ((double)end.millitm - (double)start.millitm) / 1000.0;

	printf("Hashrate: %s hashes/s | Duration: %.2f seconds | Threads: %d\n", fmt_num((size_t)(stride / seconds)).c_str(), seconds, grid_size * block_size);

	if (true == h_found) {
		char hex_result[SHA_SIZE * 2 + 1]{};
		for (int offset = 0; offset < SHA_SIZE; offset++) {
			sprintf((hex_result + (2 * offset)), "%02x", h_digest[offset] & 0xff);
		}
		printf("Nonce: %lld. Digest: %s\n", h_nonce, hex_result);
	}
	else {
		printf("Could not find nonce such that the digest ends in %d zeros\n", ZEROS_TO_FIND);
	}

Error:
	free(h_digest);
	hipFree(d_nonce);
	hipFree(d_digest);

	return status;
}